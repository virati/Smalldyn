#include "hip/hip_runtime.h"
#ifndef __SMALL_KERNS_C__
#define __SMALL_KERNS_C__

#include <stdio.h>
#include <math.h>
#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "small_kernels.cuh"
#include "smalldyn.h"

extern "C" {

__global__ void d_diff(uint N, float d_t, coord *pos_array) {
	uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= N) return;          // handle case when no. of particles not multiple of block size
    
    volatile float4 pos_data = make_float4(pos_array[index].x,pos_array[index].y, pos_array[index].z, 1.0f);
    float3 pos = make_float3(pos_data.x, pos_data.y, pos_data.z);
    
    //Generate Displacement coord
    pos += make_float3(0.1,0.1,0.1);
    
    pos_array[index].x = pos_data.x;
	pos_array[index].y = pos_data.y;
    pos_array[index].z = pos_data.z;
}

}

#endif
