#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>

#include "smallsys.h"
#include "smallsys.cuh"
#include "small_kernels.cu"

extern "C" {

void cuda_init(int argc, char **argv) {
    hipSetDevice( cutGetMaxGflopsDeviceId() );
}

void allocateArray(void **ptr, size_t size) {
    cutilSafeCall(hipMalloc(ptr, size));
}

void freeArray(void *devPtr) {
    cutilSafeCall(hipFree(devPtr));
}

void diffuse(uint num_total_particles, float time_step, coord *h_pos) {
	uint numT, numB;
	numT = 10;
	numB = 10;
	printf("yes");
	//d_diff<<<numT,numB>>>(num_total_particles, time_step, h_pos);
}

}
